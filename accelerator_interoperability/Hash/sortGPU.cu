#include <stdio.h>
#include <hiprand.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system_error.h>

// Fill d_buffer with num random numbers
extern "C" void fill_rand(float *d_buffer, int num, void *stream)
{
  hiprandGenerator_t gen;
  int status;

  // Create generator
  status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set CUDA stream
  status |= hiprandSetStream(gen, (hipStream_t)stream);

  // Set seed
  status |= hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

  // Generate num random numbers
  status |= hiprandGenerateUniform(gen, d_buffer, num);

  // Cleanup generator
  status |= hiprandDestroyGenerator(gen);

  if (status != HIPRAND_STATUS_SUCCESS) {
      printf ("hiprand failure!\n");
      exit (EXIT_FAILURE);
  }
}

// Sort key value pairs
extern "C" void sort(int *keys, int *values, int num, void *stream)
{
    try {
        // Sort keys AND values array by key
        thrust::sort_by_key(thrust::cuda::par.on((hipStream_t)stream),
                            keys, keys + num, values);
    }
    catch(thrust::system_error &e) {
        std::cerr << "Error sorting with Thrust: " << e.what() << std::endl;
        exit (EXIT_FAILURE);
    }
}
