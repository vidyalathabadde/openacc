
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Declared extern "C" to disable C++ name mangling
extern "C" void launchCUFFT(float *d_data, int n, void *stream)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftSetStream(plan, (hipStream_t)stream);
    hipfftExecC2C(plan, (hipfftComplex*)d_data, (hipfftComplex*)d_data,HIPFFT_FORWARD);
    hipfftDestroy(plan);
}
